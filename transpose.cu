#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <math.h>
#include "./include/library.h"
#include "./include/hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>

#define NDEVICE 2
#define NFILES 8
#define TIMER_DEF     struct timeval temp_1, temp_2
#define TIMER_START   gettimeofday(&temp_1, (struct timezone*)0)
#define TIMER_STOP    gettimeofday(&temp_2, (struct timezone*)0)
#define TIMER_ELAPSED ((temp_2.tv_sec-temp_1.tv_sec)+(temp_2.tv_usec-temp_1.tv_usec)/1000000.0)

int main(int argc, char *argv[]) {
    //Initialize all the stuff we need
    srand(time(NULL));
    int power = strtol(argv[1], NULL, 10);
    long number = pow(2, power);
    int gridsize = 56;
    int blocksize = 1024;

    printf("==============================================================\n");
    printf("STATS OF MY PROBLEM\n");
    printf("block size = %d \n", blocksize);
    printf("grid size = %d \n", gridsize);
    dim3 block_size(blocksize, blocksize, 1);
    dim3 grid_size(blocksize, blocksize, 1);
    printf("%d: block_size = (%d, %d), grid_size = (%d, %d)\n", __LINE__, block_size.x, block_size.y, grid_size.x, grid_size.y);
    int sharedMemSize = sizeof(dtype) * block_size.x * block_size.y * 2;

    //Print device properties for my laptop
    // FILE *file = fopen("deviceProperties.txt", "r");
    // if (file == NULL) {
    //     printf("Error opening file!\n");
    //     return 1;
    // }

    //Print device properties for unitn cluster
    FILE *file = fopen("warp.txt", "r");
    if (file == NULL) {
        printf("Error opening file!\n");
        return 1;
    }
    
    printf("==============================================================\n");
    printf("DEVICE PROPERTIES\n");
    char ch;
    while ((ch = fgetc(file)) != EOF) {
        printf("%c", ch);
    }
    fclose(file);

    //Prepare our output files changing block and grid size
    FILE *csvtime[NFILES];
    char filename[56];

    for (int i = 0; i < NFILES / 2; i++) {
        sprintf(filename, "output/timeShared%d.csv", i);
        csvtime[i] = fopen(filename, "w");
        if (csvtime[i] == NULL) {
            printf("Error opening file!\n");
            return 1;
        }

        sprintf(filename, "output/timeGlobal%d.csv", i);
        csvtime[i + NFILES / 2] = fopen(filename, "w");
        if (csvtime[i + NFILES / 2] == NULL) {
            printf("Error opening file!\n");
            return 1;
        }
    }

    for (int i = 0; i < NFILES; i++) {
        fprintf(csvtime[i], "Time,Dimensions\n");
    }
    
    //Prepare our iterations and preload kernel
    dummyKernel<<<gridsize, blocksize>>>();
    long long tries = 1 << 10;
    TIMER_DEF;
    float times[NDEVICE] = {0};

    //tries loop
    for (int count = 0; count < tries; count++) {
        hipStream_t stream;
        checkCudaErrors(hipStreamCreate(&stream));
        dtype *matrix = NULL, *transpose = NULL, *transposeShared = NULL;
        matrixInitialize(number, number, &matrix, &transpose, &transposeShared);

        //Check validity
        if (matrix == NULL || transposeShared == NULL || transpose == NULL) {
            printf("Memory allocation failed\n");
            return 1;
        }

        //Assign random values to matrices
        for (int i = 0; i < number * number; i++) {
            matrix[i] = randomf();
        }
   
        blocksize = 1024;
        gridsize = 56;
        //Matrix block transpose
        for (int k = 0; k < NFILES / 2; k++) {
            TIMER_START;
            transposeSharedMatrix<<<gridsize, blocksize, sharedMemSize, stream>>>(matrix, transposeShared, number, number);
            checkCudaErrors(hipGetLastError());
            TIMER_STOP;
            times[0] += TIMER_ELAPSED;
            fprintf(csvtime[k], "%f,%ld\n", TIMER_ELAPSED, number);
            blocksize = blocksize / 2;
            gridsize = gridsize / 2;
        }
        
        blocksize = 1024;
        gridsize = 56;
        //Matrix normal transpose
        for (int k = 0; k < NFILES / 2; k++) {
            TIMER_START;
            transposeGlobalMatrix<<<gridsize, blocksize, sharedMemSize, stream>>>(matrix, transpose, number, number);
            checkCudaErrors(hipGetLastError());
            TIMER_STOP;
            times[1] += TIMER_ELAPSED;
            fprintf(csvtime[k + NFILES / 2], "%f,%ld\n", TIMER_ELAPSED, number); 
            blocksize = blocksize / 2;
            gridsize = gridsize / 2;
        }

        //Lines for debug purposes
        //printMatrix(matrix, number, number, "Matrix");
        //printMatrix(transpose, number, number, "transpose with global memory");
        //printMatrix(transposeShared, number, number, "transpose with shared memory");
        

        matrixDestroyer(matrix, transpose, transposeShared);
        checkCudaErrors(hipStreamDestroy(stream));
        number = number + 1;
    }      

    for (int i = 0; i < NFILES; i++) {
        fclose(csvtime[i]);
    }

    printf("==============================================================\n");
    printf("STATS\n");
    printf("Global Matrix Transpose Effective Bandwidth(GB/s): %f\n", (2 * number * number * sizeof(dtype)) / (1e9 * times[1]));
    printf("Shared Matrix Transpose Effective Bandwidth(GB/s): %f\n", (2 * number * number * sizeof(dtype)) / (1e9 * times[0]));
    
    return 0;
}
