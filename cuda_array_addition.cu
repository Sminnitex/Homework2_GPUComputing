
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

float randomf();

__global__ void add(int n, float *x, float *y) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
   if(tid < n){
        y[tid] = x[tid] + y[tid];
   }
}

int main(int argc, char *argv[]){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 

    srand(time(NULL));
    long number = strtol(argv[1], NULL, 10);

    float *array;
    float *array2;

    hipMallocManaged(&array, number * sizeof(float));
    hipMallocManaged(&array2, number * sizeof(float));

    for (int i = 0; i < number; i++){
        array[i] = randomf();
        array2[i] = randomf();
    }
    int numBlocks = (number + 7) / 8;

    hipEventRecord(start);
    add<<<numBlocks, 32>>>(number, array, array2);   
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop); 

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < number; i++){
        maxError = fmax(maxError, fabs(array2[i]-3.0f));
    }
        
    std::cout << "Max error: " << maxError << std::endl; 


    float time = 0;    
    hipEventElapsedTime(&time, start, stop); 
    printf("Time spent = %f milliseconds\n", time);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipFree(array);
    hipFree(array2);

    return 0;
}

float randomf(){
    return (float) rand() / (float) RAND_MAX;   
}