#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include "./include/hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define TIMER_DEF     struct timeval temp_1, temp_2

#define TIMER_START   gettimeofday(&temp_1, (struct timezone*)0)

#define TIMER_STOP    gettimeofday(&temp_2, (struct timezone*)0)

#define TIMER_ELAPSED ((temp_2.tv_sec-temp_1.tv_sec)+(temp_2.tv_usec-temp_1.tv_usec)/1000000.0)

#define DBG_CHECK if (verbose) { printf("DBG_CHECK: file %s at line %d\n", __FILE__, __LINE__ ); }
#define DEBUG  // without debug (with random imputs) the kernel does not work

#define NPROBS 5
#define STR(s) #s
#define XSTR(s) STR(s)
#define dtype float

#define PRINT_MATRIX(A, N, M, ST ) {  \
      int i, j;  \
      printf("%s:\n", ( ST ));  \
      for (i=0; i< ( N ); i++) {  \
        printf("\t");  \
        for (j=0; j< ( M ); j++)  \
          printf("%6.3f ", A[i*( M ) + j]);  \
        printf("\n");  \
      }  \
      printf("\n\n");  \
}


float matrix_error (int n, int m, const dtype* A, const dtype* B) {
  int i, j;
  dtype error = (dtype)0;
  for (i = 0; i < n; i++)
    for (j = 0; j < m; j++)
      error += fabs(B[i*m + j] - A[i*m + j]);

  return(error);
}

#define BLOCKSIZE 64    // sgemm_global_memory_coalescing, sgemm_shared_memory_cache_blocking
#define BLOCKEDGE(R) ((BLOCKSIZE)/(R))
#define CEIL_DIV( N, D ) ((( N ) % ( D )) == 0) ? (( N )/( D )) : ((( N )/( D ))+1)

int verbose;

__global__ void naive_kernel(int N, int M, const dtype *A, const dtype *B, dtype *C) {
    // compute position in C that this thread is responsible for
    const uint tid = blockIdx.x * blockDim.x + threadIdx.x;
    int my_col = tid % M;
    int my_row = tid / M;
    extern __shared__ dtype sharedData[];

    if (tid < N*M) {
            C[my_row * M + my_col] = A[my_row * M + my_col] + B[my_row * M + my_col];
    }
}

__global__ void kernel_1(int N, int M, const dtype *A, const dtype *B, dtype *C) {
    // compute position in C that this thread is responsible for
    const uint tid = blockIdx.x * blockDim.x + threadIdx.x;
    int my_col = tid % M;
    int my_row = tid / M;

      /* |========================================| */
      /* |         Put here your kernels          | */
      /* |========================================| */

    extern __shared__ dtype sharedData[];
    dtype* sharedA = sharedData;
    dtype* sharedB = sharedData + blockDim.x * blockDim.y;
    int index = threadIdx.y * blockDim.x + threadIdx.x;


    if (tid < N*M) {
      sharedA[index] = A[my_row * M + my_col];
      sharedB[index] = B[my_row * M + my_col];
    }

    __syncthreads();

    if (tid < N*M) {
      C[my_row * M + my_col] = sharedA[index] + sharedB[index];
    } 
}

__global__ void kernel_2(int N, int M, const dtype *A, const dtype *B, dtype *C) {
    // compute position in C that this thread is responsible for
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

      /* |========================================| */
      /* |         Put here your kernels          | */
      /* |========================================| */
    extern __shared__ dtype sharedData[];

    int index = y * N + x;
    if (x < N && y < M) {
      C[index] = A[index] + B[index];
    }   
}

__global__ void kernel_3(int N, int M, const dtype *A, const dtype *B, dtype *C) {
    // compute position in C that this thread is responsible for
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

      /* |========================================| */
      /* |         Put here your kernels          | */
      /* |========================================| */

    extern __shared__ dtype sharedData[];
    dtype* sharedA = sharedData;
    dtype* sharedB = sharedData + blockDim.x * blockDim.y;

    int sharedIndex = threadIdx.y * blockDim.x + threadIdx.x;
    int index = y * N + x;

    if (x < N && y < M) {
      sharedA[sharedIndex] = A[index];
      sharedB[sharedIndex] = B[index];
    }   

    __syncthreads();

    if (x < N && y < M) {
      C[index] = sharedA[sharedIndex] + sharedB[sharedIndex];
    }   
}

dtype* execute_kernel (int n, int m, dtype* A, dtype* B, void (*kernel)(int, int, const dtype*, const dtype*, dtype*), int blk_ratio, float* Bandwidth, float* CompTime, double* Flops) {
    int grd_sizeX, grd_sizeY;
    int blk_sizeX, blk_sizeY;

    DBG_CHECK
    if ( BLOCKSIZE % blk_ratio != 0) {
        fprintf(stderr, "Error: BLOCKSIZE (%d) is not divisible for blk_ratio (%d)\n", BLOCKSIZE, blk_ratio);
        exit(__LINE__);
    }

    DBG_CHECK
    // ---------------------------------
    char sw = (kernel == naive_kernel) ? '0' : ((kernel == kernel_1) ? '1' : ((kernel == kernel_2) ? '2' : '3' ));
    switch (sw) {
        case '0' :
            blk_sizeX = BLOCKSIZE;
            grd_sizeX = CEIL_DIV(n*m, BLOCKSIZE);
            blk_sizeY = 1;
            grd_sizeY = 1;
            break;
        case '1' :
            /* Modify here your kernel launch dimension */
            blk_sizeX = BLOCKSIZE;
            grd_sizeX = CEIL_DIV(n, BLOCKSIZE);
            blk_sizeY = 1;
            grd_sizeY = 1;
            break;
        case '2' :
            /* Modify here your kernel launch dimension */
            blk_sizeX = BLOCKSIZE;
            grd_sizeX = CEIL_DIV(n, BLOCKSIZE);
            blk_sizeY = 1;
            grd_sizeY = 1;
            break;
        case '3' :
            /* Modify here your kernel launch dimension */
            blk_sizeX = BLOCKSIZE;
            grd_sizeX = CEIL_DIV(n, BLOCKSIZE);
            blk_sizeY = 1;
            grd_sizeY = 1;
            break;
    }
    // ---------------------------------

    DBG_CHECK
    // ------------------- allocating GPU vectors ----------------------
    dtype *dev_A, *dev_B, *dev_C;
    checkCudaErrors( hipMallocManaged(&dev_A, n*m*sizeof(dtype)) );
    checkCudaErrors( hipMallocManaged(&dev_B, n*m*sizeof(dtype)) );
    checkCudaErrors( hipMallocManaged(&dev_C, n*m*sizeof(dtype)) );
    size_t bandwidth_numerator = ( n * m ) * sizeof(dtype);

    DBG_CHECK
    hipStream_t stream;
    checkCudaErrors(hipStreamCreate(&stream));
    // ----------------- copy date from host to device -----------------
    //checkCudaErrors( hipMemcpy(dev_A, A, n*m*sizeof(dtype), hipMemcpyHostToDevice) );
    //checkCudaErrors( hipMemcpy(dev_B, B, n*m*sizeof(dtype), hipMemcpyHostToDevice) );
    //checkCudaErrors( hipMemset(dev_C, 0, n*m*sizeof(dtype)) );

    DBG_CHECK
    // ---------- compute GPU_tmp_b with the reduction kernel ----------
    TIMER_DEF;
    TIMER_START;

    {
        dim3 block_size(blk_sizeX, blk_sizeY, 1);
        dim3 grid_size(grd_sizeX, grd_sizeY, 1);
        printf("%d: block_size = (%d, %d), grid_size = (%d, %d)\n", __LINE__, block_size.x, block_size.y, grid_size.x, grid_size.y);
        int sharedMemSize = sizeof(dtype) * block_size.x * block_size.y * 2;
        kernel<<<grid_size, block_size, sharedMemSize, stream>>>(n, m, (const dtype*)dev_A, (const dtype*)dev_B, dev_C);
    }

    checkCudaErrors( hipDeviceSynchronize() );
    TIMER_STOP;
    *CompTime += TIMER_ELAPSED;
    *Bandwidth = bandwidth_numerator / ((*CompTime)*1e+9);
    *Flops  = ( n * m ) / ( (*CompTime) * 1e+9 );

    DBG_CHECK
    // --------------- copy results from device to host ----------------

    dtype *GPU_C;
    checkCudaErrors( hipMallocManaged(&GPU_C, n*m*sizeof(dtype)) );
    //checkCudaErrors( hipMemcpy(GPU_C, dev_C, n*m*sizeof(dtype), hipMemcpyDeviceToHost) );

    if (verbose > 0)
        PRINT_MATRIX(GPU_C, n, m, "GPU_C form execute_kernel")

    DBG_CHECK
    checkCudaErrors( hipFree(dev_A) );
    checkCudaErrors( hipFree(dev_B) );
    checkCudaErrors( hipFree(dev_C) );
    checkCudaErrors(hipStreamDestroy(stream));

    DBG_CHECK
    return(GPU_C);
}

void usage(char* bin_name, int exit_faulier) {
    fprintf(stderr, "Usage: %s -n <n> -m <m> [-c] [-v]\n", bin_name);
    fprintf(stderr, "Where mandatory inputs are:\n");
    fprintf(stderr, "\t-n\trepresents rows number of the two matrices\n");
    fprintf(stderr, "\t-m\trepresents columns number of the two matrices\n\n");

    fprintf(stderr, "And optional inputs are:\n");
    fprintf(stderr, "\t-c\tif provided it enables CPU compare computation\n");
    fprintf(stderr, "\t-v\tif provided it enables verbose prints\n\n");
    exit(exit_faulier);
}

int main(int argc, char *argv[]) {

    printf("====================================== Problem computations ======================================\n");
    // =========================================== Set-up the problem ============================================

    int p=0;
    int n, m;
    char input;
    int cpuOn_flag = 0;
    while((input = getopt(argc, argv, "n:m:cv")) != EOF) {
#define CHECKRTYPE(exitval,opt) {                                   \
    if (exitval == gread) prexit("Unexpected option -%c!\n", opt);  \
        else gread = !exitval;                                      \
    }
		switch (input) {
			//BC approx  c param is the costanst used in Bader stopping cretierion
			case 'n' :
                sscanf(optarg, "%d", &n);
                if (n <= 0) {
                    fprintf(stderr, "Error: n value must be a positive integer (%d provided)\n", n);
                    usage(argv[0], __LINE__);
                } else {
                    p |= 1;
                }
                break;
            case 'm' :
                sscanf(optarg, "%d", &m);
                if (m <= 0) {
                    fprintf(stderr, "Error: m value must be a positive integer (%d provided)\n", n);
                    usage(argv[0], __LINE__);
                } else {
                    p |= 2;
                }
                break;
            case 'c' :
                    cpuOn_flag = 1;
                break;
			case 'v' :
					verbose = 1;
					break;
			case 'h':
                usage(argv[0], __LINE__);
			case '?':
                fprintf(stderr, "Error: unrecognized parameter (%c)\n\n", input);
                usage(argv[0], __LINE__);
		}
#undef CHECKRTYPE
	}

	if (p != 3 ) {
        fprintf(stderr, "Error: -n and -m parameters are mandatory\n\n");
        usage(argv[0], __LINE__);
    }

  // ---------------- set-up the problem size -------------------



//   printf("e = %d --> n = k = m = 2^(e/2) = %d\n", e, n);
//   printf("alpha = %f, beta = %f\n", alpha, beta);
  printf("CPU_ON = %d\n", cpuOn_flag);
  printf("verbose = %d\n", verbose);
  printf("dtype = %s\n", XSTR(dtype));

  // ======================================== Get the device properties ========================================
  printf("======================================= Device properties ========================================\n");

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  int dev;
  for (dev = 0; dev < deviceCount; ++dev) {
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

    printf("  Memory Clock rate:                             %.0f Mhz\n",
           deviceProp.memoryClockRate * 1e-3f);

    printf("  Memory Bus Width:                              %d bit\n",
           deviceProp.memoryBusWidth);

    printf("  Peak Memory Bandwidth:                     %7.3f GB/s\n",
           2.0*deviceProp.memoryClockRate*(deviceProp.memoryBusWidth/8)/1.0e6);

    printf("  (%03d) Multiprocessors, (%03d) CUDA Cores/MP:    %d CUDA Cores\n",
           deviceProp.multiProcessorCount,
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
               deviceProp.multiProcessorCount);

    printf("  Peak Arithmetic Intensity:                     %7.3f GFLOPS/s\n",
           2.0*deviceProp.memoryClockRate*(_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
               deviceProp.multiProcessorCount)/1.0e6);

  }

  // ------------------ set-up the timers ---------------------

  TIMER_DEF;
  const char* lables[NPROBS] = {"CPU check", "Naive Kernel", "Kernel 1", "Kernel 2", "Kernel 3"};
  float errors[NPROBS], Times[NPROBS], Bandwidths[NPROBS], error;
  double Flops[NPROBS];
  for (int i=0; i<NPROBS; i++) {
    Bandwidths[i] = 0.0;
    errors[i] = -1.0;
    Flops[i] = 0.0;
    Times[i] = 0.0;
  }


  DBG_CHECK
  // ------------------- set-up the problem -------------------

  dtype *A, *B, *GPU_C, *CPU_C;
  checkCudaErrors( hipMallocManaged(&A, n*m*sizeof(dtype)) );
  checkCudaErrors( hipMallocManaged(&B, n*m*sizeof(dtype)) );
  checkCudaErrors( hipMallocManaged(&CPU_C, n*m*sizeof(dtype)) );
  checkCudaErrors( hipMallocManaged(&GPU_C, n*m*sizeof(dtype)) );

  time_t t;
  srand((unsigned) time(&t));


  for (int i=0; i<(n*m); i++) {
    A[i] = ((dtype)(i/m)/(dtype)m) + 1.0f;
    B[i] = (dtype)(1);
  }

#ifdef DEBUG
  if (verbose > 0) {
    PRINT_MATRIX(A, n, m, "A")
    PRINT_MATRIX(B, n, m, "B")
  }
#endif
  // ======================================== Running the computations =========================================

  /* [ ... ]
   */

  DBG_CHECK
  // ========================== CPU computation =========================
  if (cpuOn_flag) {

    TIMER_START;
    for (int i=0; i<n; i++)
      for (int j=0; j<m; j++)
        CPU_C[i*m +j] = A[i*m + j] + B[i*m + j];
    TIMER_STOP;

    Times[0] = TIMER_ELAPSED;
    errors[0] = 0.0f;
    Bandwidths[0] = -1.0f;
    Flops[0]  = (n*m) / (Times[0]*1e+9);

    if (verbose > 0)
      PRINT_MATRIX(CPU_C, n, m, "CPU_C")

    printf("CPU time: %lf\n", Times[0]);

  } else {
    Times[0] = -1.0f;
    errors[0] = -1.0f;
    Bandwidths[0] = -1.0f;
    Flops[0] = -1.0f;
  }

  DBG_CHECK
  // =========================== GPU naive Kernel ===========================
  printf("=========================== GPU naive Kernel ===========================\n");

  DBG_CHECK
  GPU_C = execute_kernel(n, m, A, B, naive_kernel, 1, &Bandwidths[1], &Times[1], &Flops[1]);

  // ------------- Compare GPU and CPU solution --------------

  (cpuOn_flag) ? (error = matrix_error(n, m, CPU_C, GPU_C)) : (error = 0.0f) ;
  errors[1] = error;

  if (verbose > 0)
    PRINT_MATRIX(GPU_C, n, m, "GPU_C")
  printf("Error equal to %lf\n", error);

  hipFree(GPU_C);

  // =========================== GPU Kernel 1 ===========================
  printf("=========================== GPU Kernel 1 ===========================\n");

  GPU_C = execute_kernel(n, m, A, B, kernel_1, 1, &Bandwidths[2], &Times[2], &Flops[2]);

  // ------------- Compare GPU and CPU solution --------------

  (cpuOn_flag) ? (error = matrix_error(n, m, CPU_C, GPU_C)) : (error = 0.0f) ;
  errors[2] = error;

  if (verbose > 0)
    PRINT_MATRIX(GPU_C, n, m, "GPU_C")

  hipFree(GPU_C);

  // =========================== GPU Kernel 2 ===========================
  printf("=========================== GPU Kernel 2 ===========================\n");

  GPU_C = execute_kernel(n, m, A, B, kernel_2, 1, &Bandwidths[3], &Times[3], &Flops[3]);

  // ------------- Compare GPU and CPU solution --------------

  (cpuOn_flag) ? (error = matrix_error(n, m, CPU_C, GPU_C)) : (error = 0.0f) ;
  errors[3] = error;

  if (verbose > 0)
    PRINT_MATRIX(GPU_C, n, m, "GPU_C")

  hipFree(GPU_C);

  // =========================== GPU Kernel 3 ===========================
  printf("=========================== GPU Kernel 3 ===========================\n");

  GPU_C = execute_kernel(n, m, A, B, kernel_3, 1, &Bandwidths[4], &Times[4], &Flops[4]);

  // ------------- Compare GPU and CPU solution --------------

  (cpuOn_flag) ? (error = matrix_error(n, m, CPU_C, GPU_C)) : (error = 0.0f) ;
  errors[4] = error;

  if (verbose > 0)
    PRINT_MATRIX(GPU_C, n, m, "GPU_C")

  hipFree(GPU_C);



  printf("\n\n");
  if (!(cpuOn_flag)) printf("CPU check not lunched!!\n");
  printf("Solution\n %9s\t%9s\t%9s\t%16s\t%16s\n", "type", "error", "time (s)", "flops (GFLOPS/s)", "bandwidth (GB/s)");
  for (int i=0; i<NPROBS; i++) {
    if ((i != 6))
      printf("%12s:\t%9.6f\t%9.6f\t%16.6lf\t%16.6f\n", lables[i], errors[i], Times[i], Flops[i], Bandwidths[i]);
  }
  printf("\n");

  printf("GPU times: n*m Kernel1_time Kernel1_flops Kernel2_time Kernel2_flops ... on stderr\n");
  fprintf(stderr, "%d, ", n*m);
  for (int i=1; i<NPROBS; i++)
    fprintf(stderr, "%f, %f, ", Times[i], Flops[i]);
  fprintf(stderr, "\n");

  return(0);
}
